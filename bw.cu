#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdlib>
#include <thread>
#include <atomic>

// Error checking macro
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

std::atomic<int> finish;
std::atomic<int> barrier;

void testBandwidth(int gpu_id, size_t dataSize, int numTransfers, int cnt, int worker_id) {
    char *hostData, *deviceData;
    hipStream_t stream;
    hipEvent_t start, stop;

    gpuErrchk(hipSetDevice(gpu_id));
    gpuErrchk(hipHostMalloc(&hostData, dataSize, hipHostMallocDefault)); // Allocate pinned host memory
    gpuErrchk(hipMalloc(&deviceData, dataSize));
    gpuErrchk(hipStreamCreate(&stream));
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    float totalMilliseconds = 0;

    barrier++;
    while (barrier < cnt);

    for (int i = 0; i < numTransfers; ++i) {
        gpuErrchk(hipEventRecord(start, stream));
        gpuErrchk(hipMemcpyAsync(deviceData, hostData, dataSize, hipMemcpyHostToDevice, stream));
        gpuErrchk(hipEventRecord(stop, stream));
        gpuErrchk(hipEventSynchronize(stop));

        float milliseconds = 0;
        gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));
        totalMilliseconds += milliseconds;
    }

    barrier--;
    while (barrier > 0);

    float averageMilliseconds = totalMilliseconds / numTransfers;
    float averageBandwidth = dataSize / (averageMilliseconds * 1e6); // GB/s

    while (finish != worker_id);
    std::cout << "GPU " << gpu_id << " Average Bandwidth: " << averageBandwidth << " GB/s" << std::endl;
    finish ++;
    // Cleanup
    hipFree(deviceData);
    hipHostFree(hostData);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <GPU_ID1> [<GPU_ID2> ...]" << std::endl;
        return -1;
    }

    const int numTransfers = 1000; // Number of transfers to average
    std::vector<std::thread> threads;
    const size_t dataSize = 1024 * 1024 * 256; // 256 MB
    finish = 0;

    for (int i = 1; i < argc; ++i) {
        int gpu_id = std::atoi(argv[i]);
        threads.emplace_back(testBandwidth, gpu_id, dataSize, numTransfers, argc-1, i-1);
    }

    // Join all threads
    for (auto &th : threads) {
        th.join();
    }

    return 0;
}
